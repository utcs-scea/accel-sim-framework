#include <iostream>
#include <hip/hip_runtime.h>

__global__ void testDependency(int *start, int *results, int n)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  int dependency_var = 0;

  // # pragma unroll
  for (int i = index; i < n; ++i)
  {
    // if (start[i] < n && i == start[i])
    // {
      dependency_var = start[i];
    // }
  }
  results[index] = dependency_var;
}

int main()
{
  unsigned n = 1024*1024;
  int results[n];

  int *d_results;
  hipMalloc(&d_results, n * sizeof(int));

  int *start, *d_start;
  hipMalloc(&d_start, n * sizeof(int));
  hipHostMalloc(&start, n * sizeof(int), hipHostMallocDefault);

  for (int i = 0; i < n; ++i)
  {
    start[i] = i;
  }

  hipMemcpy(d_start, start, n * sizeof(unsigned), hipMemcpyHostToDevice);

  testDependency<<<1, 1>>>(d_start, d_results, n);

  hipMemcpy(results, d_results, n * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_results);

  for (int i = 0; i < n / 1024; ++i)
  {
    std::cout << "Result[" << i << "]: " << results[i] << std::endl;
  }

  return 0;
}